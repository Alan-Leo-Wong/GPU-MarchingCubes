#include "hip/hip_runtime.h"
/*
 * @Author: Lei Wang leiw1006@gmail.com
 * @Date: 2023-03-18 11:32:15
 * @LastEditors: Lei Wang
 * @LastEditTime: 2023-04-18 23:01:05
 * @FilePath: \GPUMarchingCubes\MarchingCubes.h
 */
#include "Define.h"
#include "LookTable.h"
#include "MarchingCubes.h"
#include "utils\String.h"
#include <chrono>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>
#include <fstream>
#include <functional>
#include <texture_types.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <vector>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

__device__ double3 MCKernel::vertexLerp(const double3 &p_0,
                                        const double3 &p_1,
                                        const double &sdf_0,
                                        const double &sdf_1,
                                        const double &isoVal) {
    if (abs(isoVal - sdf_0) < 1e-6)
        return p_0;
    if (abs(isoVal - sdf_1) < 1e-6)
        return p_1;
    if (abs(sdf_1 - sdf_0) < 1e-6)
        return p_0;

    double t = (isoVal - sdf_0) / (sdf_1 - sdf_0);
    double3 lerp_p;
    lerp_p.x = lerp(p_0.x, p_1.x, t);
    lerp_p.y = lerp(p_0.y, p_1.y, t);
    lerp_p.z = lerp(p_0.z, p_1.z, t);
    return lerp_p;
}

__device__ double MCKernel::computeSDF(double3 pos) {
    // here is a sphere function example
    return pos.x * pos.x + pos.y * pos.y + pos.z * pos.z;
}

__device__ uint3 MCKernel::getVoxelShift(const uint &index,
                                         const uint3 &d_res) {
    // TODO
    uint x = index % d_res.x;
    uint y = index % (d_res.x * d_res.y) / d_res.x;
    uint z = index / (d_res.x * d_res.y);
    return make_uint3(x, y, z);
}

__device__ bool isNeedComputeSDF = true;

/**
 * @brief Calculates the SDF value for each voxel and determines its distribution.
 *
 * @param nVoxels          Total number of voxels = res_x * res_y * res_z
 * @param voxelSize        Size of each voxel
 * @param d_isoVal         Isosurface value
 * @param d_origin         Origin coordinates of the initial region where the MC algorithm is executed
 * @param d_res            Resolution
 * @param d_nVoxelVerts    Number of points that should be contained in each voxel after cube index mapping
 * @param d_VoxelCubeIndex Cube index corresponding to the SDF distribution within each voxel
 * @param d_voxelSDF       SDF values of the eight vertices of each voxel
 * @param d_isValidVoxel   Determines whether each voxel is a valid voxel
 */
__global__ void MCKernel::determineVoxelKernel(
        const uint nVoxels, const double *d_isoVal, const double3 *d_voxelSize,
        const double3 *d_origin, const uint3 *d_res,
        const hipTextureObject_t nVertsTex, uint *d_nVoxelVerts,
        uint *d_voxelCubeIndex, double *d_voxelSDF, uint *d_isValidVoxel) {
    uint bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint tid = bid * blockDim.x + threadIdx.x;

    if (tid < nVoxels) {
        double isoVal = *d_isoVal;

        uint3 voxelShift = getVoxelShift(tid, *d_res);
        double3 origin = *d_origin;
        double3 voxelSize = *d_voxelSize;
        double3 voxelPos; // the origin coordinate of the voxel

        voxelPos.x = origin.x + voxelShift.x * voxelSize.x;
        voxelPos.y = origin.y + voxelShift.y * voxelSize.y;
        voxelPos.z = origin.z + voxelShift.z * voxelSize.z;

        double3 corners[8];
        corners[0] = voxelPos;
        corners[1] = voxelPos + make_double3(0, voxelSize.y, 0);
        corners[2] = voxelPos + make_double3(voxelSize.x, voxelSize.y, 0);
        corners[3] = voxelPos + make_double3(voxelSize.x, 0, 0);
        corners[4] = voxelPos + make_double3(0, 0, voxelSize.z);
        corners[5] = voxelPos + make_double3(0, voxelSize.y, voxelSize.z);
        corners[6] = voxelPos + make_double3(voxelSize.x, voxelSize.y, voxelSize.z);
        corners[7] = voxelPos + make_double3(voxelSize.x, 0, voxelSize.z);

        double sdf[8];
        if (isNeedComputeSDF) {
            for (int i = 0; i < 8; ++i) {
                sdf[i] = computeSDF(corners[i]);
                d_voxelSDF[tid * 8 + i] = sdf[i];
            }
        } else {
            for (int i = 0; i < 8; ++i) {
                sdf[i] = d_voxelSDF[tid * 8 + i];
#ifndef NDEBUG
                if (tid == 0) {
                  printf("sdf = %lf\n", sdf[i]);
                }
#endif
            }
        }

        int cubeIndex = 0;
        cubeIndex = (uint(sdf[0] < isoVal)) | (uint(sdf[1] < isoVal) << 1) |
                    (uint(sdf[2] < isoVal) << 2) | (uint(sdf[3] < isoVal) << 3) |
                    (uint(sdf[4] < isoVal) << 4) | (uint(sdf[5] < isoVal) << 5) |
                    (uint(sdf[6] < isoVal) << 6) | (uint(sdf[7] < isoVal) << 7);

        int nVerts = tex1Dfetch<int>(nVertsTex, cubeIndex);
        d_nVoxelVerts[tid] = nVerts;
        d_isValidVoxel[tid] = nVerts > 0;
        d_voxelCubeIndex[tid] = cubeIndex;
    }
}

/**
 * @brief Compact voxel array
 *
 * @param nVoxels               Total number of voxels = res_x * res_y * res_z
 * @param d_isValidVoxel        Determines whether each voxel is a valid voxel
 * @param d_nValidVoxelsScan    Exclusive sum of d_isValidVoxel
 * @param d_compactedVoxelArray Output
 */
__global__ void MCKernel::compactVoxels(const uint nVoxels,
                                        const uint *d_isValidVoxel,
                                        const uint *d_nValidVoxelsScan,
                                        uint *d_compactedVoxelArray) {
    uint bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint tid = bid * blockDim.x + threadIdx.x;

    if (tid < nVoxels && d_isValidVoxel[tid])
        d_compactedVoxelArray[d_nValidVoxelsScan[tid]] = tid;
}

/**
 * @brief Converts the SDF distribution of each voxel into a mesh
 *
 * @param maxVerts              Maximum possible number of points included by the MC algorithm
 * @param nValidVoxels          Total number of valid voxels = res_x * res_y * res_z
 * @param voxelSize             Size of each voxel
 * @param d_isoVal              Isosurface value
 * @param d_origin              Origin coordinates of the initial region where the MC algorithm is executed
 * @param d_res                 Resolution
 * @param d_compactedVoxelArray Array of voxels with invalid entries removed
 * @param d_nVoxelVerts         Number of points that should be contained in each voxel after cube index mapping
 * @param d_voxelCubeIndex      Cube index corresponding to the SDF distribution within each voxel
 * @param d_voxelSDF            SDF values of the eight vertices of each voxel
 * @param d_nVertsScanned       Prefix sum of the point count in all valid voxels
 * @param d_triPoints           Output, stores the position of all points in the actual mesh
 */
__global__ void MCKernel::voxelToMeshKernel(
        const uint nValidVoxels, const int maxVerts, const double *d_isoVal,
        const double3 *d_voxelSize, const double3 *d_origin, const uint3 *d_res,
        const uint *d_compactedVoxelArray, const hipTextureObject_t nVertsTex,
        const hipTextureObject_t triTex, uint *d_voxelCubeIndex,
        double *d_voxelSDF, uint *d_nVertsScanned, double3 *d_triPoints) {
    uint bid = blockIdx.y * gridDim.x + blockIdx.x;
    uint tid = bid * blockDim.x + threadIdx.x;

    if (tid < nValidVoxels) {
        uint voxelIdx = d_compactedVoxelArray[tid];

        double isoVal = *d_isoVal;

        uint3 voxelShift = getVoxelShift(voxelIdx, *d_res);
        double3 voxelPos; // voxel 原点坐标
        double3 voxelSize = *d_voxelSize;

        voxelPos.x = voxelShift.x * voxelSize.x;
        voxelPos.y = voxelShift.y * voxelSize.y;
        voxelPos.z = voxelShift.z * voxelSize.z;
        voxelPos += (*d_origin);

        uint cubeIndex = d_voxelCubeIndex[voxelIdx];
        double sdf[8];
        for (int i = 0; i < 8; ++i)
            sdf[i] = d_voxelSDF[voxelIdx * 8 + i];

        double3 corners[8];
        corners[0] = voxelPos;
        corners[1] = voxelPos + make_double3(0, voxelSize.y, 0);
        corners[2] = voxelPos + make_double3(voxelSize.x, voxelSize.y, 0);
        corners[3] = voxelPos + make_double3(voxelSize.x, 0, 0);
        corners[4] = voxelPos + make_double3(0, 0, voxelSize.z);
        corners[5] = voxelPos + make_double3(0, voxelSize.y, voxelSize.z);
        corners[6] = voxelPos + make_double3(voxelSize.x, voxelSize.y, voxelSize.z);
        corners[7] = voxelPos + make_double3(voxelSize.x, 0, voxelSize.z);

        // To prevent thread divergence, calculate the interpolation points on all 12 edges once,
        // since the final triangle arrangement is also determined by the cube index.
        double3 triVerts[12];
        triVerts[0] = vertexLerp(corners[0], corners[1], sdf[0], sdf[1], isoVal);
        triVerts[1] = vertexLerp(corners[1], corners[2], sdf[1], sdf[2], isoVal);
        triVerts[2] = vertexLerp(corners[2], corners[3], sdf[2], sdf[3], isoVal);
        triVerts[3] = vertexLerp(corners[3], corners[0], sdf[3], sdf[0], isoVal);

        triVerts[4] = vertexLerp(corners[4], corners[5], sdf[4], sdf[5], isoVal);
        triVerts[5] = vertexLerp(corners[5], corners[6], sdf[5], sdf[6], isoVal);
        triVerts[6] = vertexLerp(corners[6], corners[7], sdf[6], sdf[7], isoVal);
        triVerts[7] = vertexLerp(corners[7], corners[4], sdf[7], sdf[4], isoVal);

        triVerts[8] = vertexLerp(corners[0], corners[4], sdf[0], sdf[4], isoVal);
        triVerts[9] = vertexLerp(corners[1], corners[5], sdf[1], sdf[5], isoVal);
        triVerts[10] = vertexLerp(corners[2], corners[6], sdf[2], sdf[6], isoVal);
        triVerts[11] = vertexLerp(corners[3], corners[7], sdf[3], sdf[7], isoVal);

        int nVerts = tex1Dfetch<int>(nVertsTex, cubeIndex);

        for (int i = 0; i < nVerts; i += 3) {
            uint triPosIndex = d_nVertsScanned[voxelIdx] + i;

            double3 triangle[3];

            int edgeIndex = tex1Dfetch<int>(triTex, (cubeIndex * 16) + i);
            triangle[0] = triVerts[edgeIndex];

            edgeIndex = tex1Dfetch<int>(triTex, (cubeIndex * 16) + i + 1);
            triangle[1] = triVerts[edgeIndex];

            edgeIndex = tex1Dfetch<int>(triTex, (cubeIndex * 16) + i + 2);
            triangle[2] = triVerts[edgeIndex];

            if (triPosIndex < maxVerts - 3) {
                d_triPoints[triPosIndex] = triangle[0];
                d_triPoints[triPosIndex + 1] = triangle[1];
                d_triPoints[triPosIndex + 2] = triangle[2];
            }
        }
    }
}

namespace MC {
    // host
    namespace {
        uint allTriVertices = 0, nValidVoxels = 0;

        double3 *h_triPoints = nullptr; // output
    } // namespace

    // device
    namespace {
        uint3 *d_res = nullptr;
        double *d_isoVal = nullptr;

        uint *d_nVoxelVertsArray = nullptr;
        uint *d_nVoxelVertsScan = nullptr;

        uint *d_isValidVoxelArray = nullptr;
        uint *d_nValidVoxelsScan = nullptr;

        double3 *d_gridOrigin = nullptr;
        double3 *d_voxelSize = nullptr;

        double *d_voxelSDF = nullptr;
        uint *d_voxelCubeIndex = nullptr;

        uint *d_compactedVoxelArray = nullptr;

        int *d_triTable = nullptr;
        int *d_nVertsTable = nullptr;

        // textures containing look-up tables
        hipTextureObject_t triTex;
        hipTextureObject_t nVertsTex;

        double3 *d_triPoints = nullptr; // output
    } // namespace
} // namespace MC

void MC::d_thrustExclusiveScan(const uint &nElems, uint *input,
                               uint *output) {
    thrust::exclusive_scan(thrust::device_ptr<uint>(input),
                           thrust::device_ptr<uint>(input + nElems),
                           thrust::device_ptr<uint>(output));
}

void MC::setTextureObject(const uint &srcSizeInBytes, int *srcDev,
                          hipTextureObject_t *texObj) {
    hipChannelFormatDesc channelDesc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    hipResourceDesc texRes;
    hipTextureDesc texDesc;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    memset(&texDesc, 0, sizeof(hipTextureDesc));

    texRes.resType = hipResourceTypeLinear;
    texRes.res.linear.devPtr = srcDev;
    texRes.res.linear.sizeInBytes = srcSizeInBytes;
    texRes.res.linear.desc = channelDesc;

    texDesc.normalizedCoords = false;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.readMode = hipReadModeElementType;

    CUDA_CHECK(hipCreateTextureObject(texObj, &texRes, &texDesc, nullptr));
}

void MC::initResources(const bool &sdfFlag, const uint3 &resolution,
                       const uint &nVoxels, const double &isoVal,
                       const double3 &gridOrigin,
                       const double3 &voxelSize, const uint &maxVerts,
                       const std::vector<double> h_voxelSDF) {
    // host
    {
        h_triPoints = (double3 *) malloc(sizeof(double3) * maxVerts);
        // printf("h_triPoints = %d\n", h_triPoints);
    }

    // device
    {
        CUDA_CHECK(hipMalloc((void **) &d_res, sizeof(uint3)));
        CUDA_CHECK(
                hipMemcpy(d_res, &resolution, sizeof(uint3), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &d_isoVal, sizeof(double)));
        CUDA_CHECK(
                hipMemcpy(d_isoVal, &isoVal, sizeof(double), hipMemcpyHostToDevice));

        CUDA_CHECK(
                hipMalloc((void **) &d_nVoxelVertsArray, sizeof(uint) * nVoxels));
        CUDA_CHECK(hipMalloc((void **) &d_nVoxelVertsScan, sizeof(uint) * nVoxels));

        CUDA_CHECK(
                hipMalloc((void **) &d_isValidVoxelArray, sizeof(uint) * nVoxels));
        CUDA_CHECK(
                hipMalloc((void **) &d_nValidVoxelsScan, sizeof(uint) * nVoxels));

        CUDA_CHECK(hipMalloc((void **) &d_gridOrigin, sizeof(double3)));
        CUDA_CHECK(hipMemcpy(d_gridOrigin, &gridOrigin, sizeof(double3),
                              hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &d_voxelSize, sizeof(double3)));
        CUDA_CHECK(hipMemcpy(d_voxelSize, &voxelSize, sizeof(double3),
                              hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &d_voxelSDF, sizeof(double) * nVoxels * 8));
        if (!sdfFlag) {
            assert(h_voxelSDF.size() >= nVoxels * 8);
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(isNeedComputeSDF), &sdfFlag, sizeof(bool)));
            CUDA_CHECK(hipMemcpy(d_voxelSDF, h_voxelSDF.data(),
                                  sizeof(double) * nVoxels * 8,
                                  hipMemcpyHostToDevice));
        }

        CUDA_CHECK(hipMalloc((void **) &d_voxelCubeIndex, sizeof(uint) * nVoxels));

        CUDA_CHECK(hipMalloc((void **) &d_triTable, sizeof(int) * 256 * 16));
        CUDA_CHECK(hipMemcpy(d_triTable, triTable, sizeof(int) * 256 * 16,
                              hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &d_nVertsTable, sizeof(int) * 256));
        CUDA_CHECK(hipMemcpy(d_nVertsTable, nVertsTable, sizeof(int) * 256,
                              hipMemcpyHostToDevice));

        // texture
        setTextureObject(256 * 16 * sizeof(int), d_triTable, &triTex);
        setTextureObject(256 * sizeof(int), d_nVertsTable, &nVertsTex);

        CUDA_CHECK(hipMalloc((void **) &d_triPoints, sizeof(double3) * maxVerts));
    }
}

void MC::freeResources() {
    // host
    { free(h_triPoints); }

    // device
    {
        CUDA_CHECK(hipFree(d_res));

        CUDA_CHECK(hipFree(d_nVoxelVertsArray));
        CUDA_CHECK(hipFree(d_nVoxelVertsScan);)

        CUDA_CHECK(hipFree(d_isValidVoxelArray));
        CUDA_CHECK(hipFree(d_nValidVoxelsScan));

        CUDA_CHECK(hipFree(d_gridOrigin));
        CUDA_CHECK(hipFree(d_voxelSize));

        CUDA_CHECK(hipFree(d_voxelSDF));
        CUDA_CHECK(hipFree(d_voxelCubeIndex));

        CUDA_CHECK(hipFree(d_triTable));
        CUDA_CHECK(hipFree(d_nVertsTable));

        // texture object
        CUDA_CHECK(hipDestroyTextureObject(triTex));
        CUDA_CHECK(hipDestroyTextureObject(nVertsTex));

        CUDA_CHECK(hipFree(d_triPoints));
    }
}

void MC::launch_determineVoxelKernel(const uint &nVoxels,
                                     const double &isoVal,
                                     const uint &maxVerts) {
    dim3 nThreads(NTHREADS, 1, 1);
    dim3 nBlocks((nVoxels + nThreads.x - 1) / nThreads.x, 1, 1);
    while (nBlocks.x > 65535) {
        nBlocks.x /= 2;
        nBlocks.y *= 2;
    }

    MCKernel::determineVoxelKernel<<<nBlocks, nThreads>>>(
            nVoxels, d_isoVal, d_voxelSize, d_gridOrigin, d_res, nVertsTex,
                    d_nVoxelVertsArray, d_voxelCubeIndex, d_voxelSDF, d_isValidVoxelArray);
    getLastCudaError("Kernel: 'determineVoxelKernel' failed!\n");
#ifndef NDEBUG
    hipDeviceSynchronize();
#endif

    d_thrustExclusiveScan(nVoxels, d_nVoxelVertsArray, d_nVoxelVertsScan);
    d_thrustExclusiveScan(nVoxels, d_isValidVoxelArray, d_nValidVoxelsScan);

    uint lastElement, lastScanElement;
    CUDA_CHECK(hipMemcpy(&lastElement, d_isValidVoxelArray + nVoxels - 1,
                          sizeof(uint), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&lastScanElement, d_nValidVoxelsScan + nVoxels - 1,
                          sizeof(uint), hipMemcpyDeviceToHost));
    nValidVoxels = lastElement + lastScanElement;
    if (nValidVoxels == 0)
        return;

    CUDA_CHECK(hipMemcpy(&lastElement, d_nVoxelVertsArray + nVoxels - 1,
                          sizeof(uint), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&lastScanElement, d_nVoxelVertsScan + nVoxels - 1,
                          sizeof(uint), hipMemcpyDeviceToHost));
    allTriVertices = lastElement + lastScanElement;
}

void MC::launch_compactVoxelsKernel(const int &nVoxels) {
    CUDA_CHECK(
            hipMalloc((void **) &d_compactedVoxelArray, sizeof(uint) * nVoxels));

    dim3 nThreads(NTHREADS, 1, 1);
    dim3 nBlocks((nVoxels + nThreads.x - 1) / nThreads.x, 1, 1);
    while (nBlocks.x > 65535) {
        nBlocks.x /= 2;
        nBlocks.y *= 2;
    }

    MCKernel::compactVoxels<<<nBlocks, nThreads>>>(
            nVoxels, d_isValidVoxelArray, d_nValidVoxelsScan, d_compactedVoxelArray);
    getLastCudaError("Kernel: 'compactVoxelsKernel' failed!\n");
}

void MC::launch_voxelToMeshKernel(const uint &maxVerts,
                                  const uint &nVoxels) {
    if(nValidVoxels == 0) return;

    dim3 nThreads(NTHREADS, 1, 1);
    dim3 nBlocks((nValidVoxels + nThreads.x - 1) / nThreads.x, 1, 1);
    while (nBlocks.x > 65535) {
        nBlocks.x /= 2;
        nBlocks.y *= 2;
    }

    MCKernel::voxelToMeshKernel<<<nBlocks, nThreads>>>(
            nValidVoxels, maxVerts, d_isoVal, d_voxelSize, d_gridOrigin, d_res,
                    d_compactedVoxelArray, nVertsTex, triTex, d_voxelCubeIndex, d_voxelSDF,
                    d_nVoxelVertsScan, d_triPoints);
    getLastCudaError("Kernel: 'voxelToMeshKernel' failed!\n");
    CUDA_CHECK(hipMemcpy(h_triPoints, d_triPoints, sizeof(double3) * maxVerts,
                          hipMemcpyDeviceToHost));
}

void MC::writeToOBJFile(const std::string &filename) {
    checkDir(filename);
    std::ofstream out(filename);
    if (!out) {
        fprintf(stderr, "IO Error: File %s could not be opened!\n",
                filename.c_str());
        return;
    }

    printf("The number of mesh's vertices = %d\n", allTriVertices);
    printf("The number of mesh's faces = %d\n", allTriVertices / 3);
    for (int i = 0; i < allTriVertices; i += 3) {
        const int faceIdx = i;

        out << "v " << h_triPoints[i].x << ' ' << h_triPoints[i].y << ' '
            << h_triPoints[i].z << '\n';
        out << "v " << h_triPoints[i + 1].x << ' ' << h_triPoints[i + 1].y << ' '
            << h_triPoints[i + 1].z << '\n';
        out << "v " << h_triPoints[i + 2].x << ' ' << h_triPoints[i + 2].y << ' '
            << h_triPoints[i + 2].z << '\n';

        out << "f " << faceIdx + 1 << ' ' << faceIdx + 2 << ' ' << faceIdx + 3
            << '\n';
    }

    out.close();
}

void MC::marching_cubes(const uint3 &resolution, const double3 &gridOrigin,
                        const double3 &voxelSize, const double &isoVal,
                        const bool &sdfFlag, const std::string &filename,
                        const std::vector<double> &h_voxelSDF) {
    uint nVoxels = resolution.x * resolution.y * resolution.z;
    uint maxVerts = nVoxels * 18;

    using namespace std::chrono;
    time_point<system_clock> start, end;

    start = system_clock::now();

    initResources(sdfFlag, resolution, nVoxels, isoVal, gridOrigin, voxelSize,
                  maxVerts, h_voxelSDF);

    launch_determineVoxelKernel(nVoxels, isoVal, maxVerts);
    if (allTriVertices == 0) {
        printf("There is no valid vertices...\n");
        return;
    }

    launch_compactVoxelsKernel(nVoxels);

    launch_voxelToMeshKernel(maxVerts, nVoxels);

    end = system_clock::now();
    duration<double> elapsed_seconds = end - start;
    std::time_t end_time = system_clock::to_time_t(end);
    std::cout << "Finished computation at " << std::ctime(&end_time)
              << "Elapsed time: " << elapsed_seconds.count() << "s\n----------\n";

    std::cout << "Write to obj..." << std::endl;
    writeToOBJFile(filename);

    freeResources();
}
